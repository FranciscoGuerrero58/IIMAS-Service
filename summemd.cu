#include "../common/book.h"
#define N 10
int main( void ) {
int a[N], b[N], c[N];
int *dev_a, *dev_b, *dev_c;
// allocate the memory on the GPU
HANDLE_ERROR( hipMalloc( (void**)&dev_a, N * sizeof(int) ) );
HANDLE_ERROR( hipMalloc( (void**)&dev_b, N * sizeof(int) ) );
HANDLE_ERROR( hipMalloc( (void**)&dev_c, N * sizeof(int) ) );
// fill the arrays 'a' and 'b' on the CPU
for (int i=0; i<N; i++) {
a[i] = -i;
b[i] = i * i;
}
HANDLE_ERROR(hipMemcpy(dev_a, a, N * sizeof(int),hipMemcpyHostToDevice));
HANDLE_ERROR(hipMemcpy(dev_b, b, N * sizeof(int),hipMemcpyHostToDevice));
add<<<N,1>>>(dev_a, dev_b, dev_c);

HANDLE_ERROR(hipMemcpy(c, dev_c, N * sizeof(int),hipMemcpyHostToDevice));
for(int i=0; i<N; i++){
	printf("%d + %d = %d\n", a[i], b[i], c[i]);
}

hipFree(dev_a);
hipFree(dev_b);
hipFree(dev_c);

return 0;
}